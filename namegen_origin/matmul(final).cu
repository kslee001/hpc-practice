#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"
#include <hip/hip_runtime.h>
#include <mpi.h>
#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }
#define MIN(a,b) ((a) > (b)) ? (b) : (a)
#define MAX_NUM_GPU 4
#define TILE_SIZE 16
int num_devices = 0;

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K) {
  __shared__ float Asub[TILE_SIZE][TILE_SIZE];
  __shared__ float Bsub[TILE_SIZE][TILE_SIZE];

  int global_row = blockDim.y * blockIdx.y + threadIdx.y;
  int global_col = blockDim.x * blockIdx.x + threadIdx.x;
  float sum = 0.0f;
  int num_tiles = (K+TILE_SIZE-1)/TILE_SIZE;

  for (int tile = 0; tile < num_tiles; ++tile){
  if ( (global_row < M) && (threadIdx.x + tile*TILE_SIZE < K) )
    Asub[threadIdx.y][threadIdx.x] = A[global_row * K + threadIdx.x + tile * TILE_SIZE];
  else
    Asub[threadIdx.y][threadIdx.x] = 0.0f;

  if ( (global_col < N) && (threadIdx.y + tile * TILE_SIZE < K ))
    Bsub[threadIdx.y][threadIdx.x] = B[(threadIdx.y + tile * TILE_SIZE) * N + global_col];
  else
    Bsub[threadIdx.y][threadIdx.x] = 0.0f;
  
  __syncthreads();
  
  for (int k = 0; k < TILE_SIZE; k++)
    sum += Asub[threadIdx.y][k] * Bsub[k][threadIdx.x];
  
  __syncthreads();
  }

  if (global_row < M && global_col < N){
  C[global_row * N + global_col] = sum;
  }
}
static int mpi_rank, mpi_world_size;

// Array of device (GPU) pointers
static float *a_d[MAX_NUM_GPU];
static float *b_d[MAX_NUM_GPU];
static float *c_d[MAX_NUM_GPU];
static int Mbegin[MAX_NUM_GPU], Mend[MAX_NUM_GPU];
static int hostBegin[4], hostEnd[4];
static MPI_Status status;

void matmul(const float *A, const float *B, float *C, int M, int N, int K) {
  // send & recv data (root -> CPUs)
  if (mpi_rank==0) {
    for (int i=1; i< mpi_world_size; i++) {
      MPI_Send((float*) A+hostBegin[i]*K, (hostEnd[i]-hostBegin[i])*K, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
      MPI_Send((float*) B, K*N, MPI_FLOAT, i, 2, MPI_COMM_WORLD);
    }
  } else {
      MPI_Recv((float*) A+hostBegin[mpi_rank]*K, (hostEnd[mpi_rank]-hostBegin[mpi_rank])*K, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, &status);
      MPI_Recv((float*) B, K*N, MPI_FLOAT, 0, 2, MPI_COMM_WORLD, &status);
  }

  MPI_Barrier(MPI_COMM_WORLD);

  // Upload A and B matrix to every GPU
  for (int i = 0; i < num_devices; i++) { //a_d (global) <- A (cpu)
    CUDA_CALL(hipMemcpy(a_d[i], A + Mbegin[i] * K,
                         (Mend[i] - Mbegin[i]) * K * sizeof(float),
                         hipMemcpyHostToDevice));
    CUDA_CALL(
        hipMemcpy(b_d[i], B, K * N * sizeof(float), hipMemcpyHostToDevice));
  }

  // Launch kernel on every GPU
  for (int i = 0; i < num_devices; i++) {
    // dim3 gridDim(ceil((Mend[i] - Mbegin[i]) /(float) TILE_SIZE), ceil(N/ (float) TILE_SIZE), 1);
    dim3 gridDim((N - 1) / TILE_SIZE + 1, (Mend[i] - Mbegin[i] - 1) / TILE_SIZE + 1, 1);
    dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);

    CUDA_CALL(hipSetDevice(i));
    // gridDim: block 개수
    // blockDim: thread 개수
    matmul_kernel<<<gridDim, blockDim>>>(a_d[i], b_d[i], c_d[i], M, N, K);
  }

  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipDeviceSynchronize());
  }

  // Download C matrix from GPUs
  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipMemcpy(C + Mbegin[i] * N, c_d[i],
                        (Mend[i] - Mbegin[i]) * N * sizeof(float),
                        hipMemcpyDeviceToHost));
  }
 
  MPI_Barrier(MPI_COMM_WORLD);

  if (mpi_rank == 0) {
    for (int i=1; i < mpi_world_size; i++) {
      MPI_Recv((float*) C+hostBegin[i]*N, (hostEnd[i]-hostBegin[i])*N, MPI_FLOAT, i, 1, MPI_COMM_WORLD, &status);
    }
  } else {
    MPI_Send((float*) C+hostBegin[mpi_rank]*N, (hostEnd[mpi_rank]-hostBegin[mpi_rank])*N, MPI_FLOAT, 0, 1, MPI_COMM_WORLD);
  }
  
  MPI_Barrier(MPI_COMM_WORLD);

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipDeviceSynchronize());
  }
  
}

void matmul_initialize(int M, int N, int K) {
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
  // setting Mbegin & Mend for each host(cpu)
  CUDA_CALL(hipGetDeviceCount(&num_devices));
  for (int i=0; i<mpi_world_size; ++i) {
    hostBegin[i] =  (M/mpi_world_size)*i;
    hostEnd[i] = (M/mpi_world_size)*(i+1);
  }
  hostEnd[mpi_world_size-1] = M;

  int portion = hostEnd[mpi_rank]-hostBegin[mpi_rank];
  for (int i = 0; i < num_devices; i++) {
    Mbegin[i] = hostBegin[mpi_rank] + (portion / num_devices) * i;
    Mend[i] = hostBegin[mpi_rank] + (portion / num_devices) * (i + 1);
  }
  Mend[num_devices - 1] = hostEnd[mpi_rank];
  MPI_Barrier(MPI_COMM_WORLD);

  // Only root process do something
  if (mpi_rank == 0) {
    printf("Using %d devices\n", num_devices);
    for (int i = 0; i < num_devices; i++) {
      hipDeviceProp_t prop;
      CUDA_CALL(hipGetDeviceProperties(&prop, i));

      // Try printing more detailed information here
      printf("GPU %d: %s\n", i, prop.name);
    }

    if (num_devices <= 0) {
      printf("No CUDA device found. Aborting\n");
      exit(1);
    }

    // Setup problem size for each GPU
    // Allocate device memory for each GPU
  }

  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipSetDevice(i));
    CUDA_CALL(hipMalloc(&a_d[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CUDA_CALL(hipMalloc(&b_d[i], K * N * sizeof(float)));
    CUDA_CALL(hipMalloc(&c_d[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }  
}

void matmul_finalize() {

  // Only root process do something
  if (mpi_rank == 0) {
    // Free all GPU memory
    for (int i = 0; i < num_devices; i++) {
      CUDA_CALL(hipFree(a_d[i]));
      CUDA_CALL(hipFree(b_d[i]));
      CUDA_CALL(hipFree(c_d[i]));
    }
  }
}