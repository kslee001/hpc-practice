#include "hip/hip_runtime.h"
#include <stdlib.h>

#include <cmath>
#include <cstdint>
#include <cstdio>
#include <cstdlib>

#include "tensor.h"
#include "uNet.h"
#include "util.h"

// Parameters for U-Net
Tensor *inc_double_conv_0_weight;
Tensor *inc_double_conv_1_weight;
Tensor *inc_double_conv_1_bias;
Tensor *inc_double_conv_3_weight;
Tensor *inc_double_conv_4_weight;
Tensor *inc_double_conv_4_bias;
Tensor *down1_maxpool_conv_1_double_conv_0_weight;
Tensor *down1_maxpool_conv_1_double_conv_1_weight;
Tensor *down1_maxpool_conv_1_double_conv_1_bias;
Tensor *down1_maxpool_conv_1_double_conv_3_weight;
Tensor *down1_maxpool_conv_1_double_conv_4_weight;
Tensor *down1_maxpool_conv_1_double_conv_4_bias;
Tensor *down2_maxpool_conv_1_double_conv_0_weight;
Tensor *down2_maxpool_conv_1_double_conv_1_weight;
Tensor *down2_maxpool_conv_1_double_conv_1_bias;
Tensor *down2_maxpool_conv_1_double_conv_3_weight;
Tensor *down2_maxpool_conv_1_double_conv_4_weight;
Tensor *down2_maxpool_conv_1_double_conv_4_bias;
Tensor *up1_up_weight;
Tensor *up1_up_bias;
Tensor *up1_conv_double_conv_0_weight;
Tensor *up1_conv_double_conv_1_weight;
Tensor *up1_conv_double_conv_1_bias;
Tensor *up1_conv_double_conv_3_weight;
Tensor *up1_conv_double_conv_4_weight;
Tensor *up1_conv_double_conv_4_bias;
Tensor *up2_up_weight;
Tensor *up2_up_bias;
Tensor *up2_conv_double_conv_0_weight;
Tensor *up2_conv_double_conv_1_weight;
Tensor *up2_conv_double_conv_1_bias;
Tensor *up2_conv_double_conv_3_weight;
Tensor *up2_conv_double_conv_4_weight;
Tensor *up2_conv_double_conv_4_bias;
Tensor *outc_conv_weight;
Tensor *outc_conv_bias;
Tensor *inc_batchnorm_0_running_mean;
Tensor *inc_batchnorm_0_running_var;
Tensor *down1_batchnorm_0_running_mean;
Tensor *down1_batchnorm_0_running_var;
Tensor *down2_batchnorm_0_running_mean;
Tensor *down2_batchnorm_0_running_var;
Tensor *up1_batchnorm_0_running_mean;
Tensor *up1_batchnorm_0_running_var;
Tensor *up2_batchnorm_0_running_mean;
Tensor *up2_batchnorm_0_running_var;
Tensor *inc_batchnorm_1_running_mean;
Tensor *inc_batchnorm_1_running_var;
Tensor *down1_batchnorm_1_running_mean;
Tensor *down1_batchnorm_1_running_var;
Tensor *down2_batchnorm_1_running_mean;
Tensor *down2_batchnorm_1_running_var;
Tensor *up1_batchnorm_1_running_mean;
Tensor *up1_batchnorm_1_running_var;
Tensor *up2_batchnorm_1_running_mean;
Tensor *up2_batchnorm_1_running_var;

// intermediate features
Tensor *inc_conv_0_output;
Tensor *inc_batchnorm_0_output;
Tensor *inc_conv_1_output;
Tensor *inc_batchnorm_1_output;
Tensor *down1_maxpool2d_0_output;
Tensor *down1_conv_0_output;
Tensor *down1_batchnorm_0_output;
Tensor *down1_conv_1_output;
Tensor *down1_batchnorm_1_output;
Tensor *down2_maxpool2d_0_output;
Tensor *down2_conv_0_output;
Tensor *down2_batchnorm_0_output;
Tensor *down2_conv_1_output;
Tensor *down2_batchnorm_1_output;
Tensor *up1_convt_0_output;
Tensor *up1_concat_0_output;
Tensor *up1_conv_0_output;
Tensor *up1_batchnorm_0_output;
Tensor *up1_conv_1_output;
Tensor *up1_batchnorm_1_output;
Tensor *up2_convt_0_output;
Tensor *up2_concat_0_output;
Tensor *up2_conv_0_output;
Tensor *up2_batchnorm_0_output;
Tensor *up2_conv_1_output;
Tensor *up2_batchnorm_1_output;
Tensor *outc_conv_0_output;

// forward declaration, prototype
void Conv2d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride, int pad, int dilation, bool has_bias);
void ReLU(Tensor *inout);
void BatchNorm2d(Tensor *input, Tensor *gamma, Tensor *beta,
                 Tensor *running_mean, Tensor *running_var, Tensor *output,
                 const float eps, const float momentum);
void ConvTranspose2d(Tensor *input, Tensor *weight, Tensor *bias,
                     Tensor *output, int stride, int pad);
void MaxPool2d(Tensor *input, Tensor *output);
void Concat(Tensor *input1, Tensor *input2, Tensor *output);
void uNet_initialize(int, int, char *);
void uNet(Tensor *, Tensor *);
void uNet_finalize();

/*
 * uNet
 * This model identifies the boundaries of the cars in an image file (input.bin)
 * and removes the background.
 */

void uNet(Tensor *inputN, Tensor *outputN, int N) {  
  Tensor *input = new Tensor({1, 3, 128, 191});   // input image tensor
  Tensor *output = new Tensor({1, 2, 128, 191});  // output image tensor (binary : segmentation)

  for (int idx = 0; idx < N; ++idx) {
    memcpy(input->buf, inputN->buf + (idx * 1 * 3 * 128 * 191),
           sizeof(float) * 1 * 3 * 128 * 191);

    // inc(n_channels, 64)
    Conv2d(input, inc_double_conv_0_weight, NULL, inc_conv_0_output, 1, 1, 1, // input, (weight, bias,) ouput,   ( stride pad dilatation ) , bias
           false);
    BatchNorm2d(inc_conv_0_output, inc_double_conv_1_weight,
                inc_double_conv_1_bias, inc_batchnorm_0_running_mean,
                inc_batchnorm_0_running_var, inc_batchnorm_0_output, 1e-5, 0.1);  
    ReLU(inc_batchnorm_0_output);
    Conv2d(inc_batchnorm_0_output, inc_double_conv_3_weight, NULL,
           inc_conv_1_output, 1, 1, 1, false);
    BatchNorm2d(inc_conv_1_output, inc_double_conv_4_weight,
                inc_double_conv_4_bias, inc_batchnorm_1_running_mean,
                inc_batchnorm_1_running_var, inc_batchnorm_1_output, 1e-5, 0.1);
    ReLU(inc_batchnorm_1_output);

    // down1(64, 128)
    MaxPool2d(inc_batchnorm_1_output, down1_maxpool2d_0_output);
    Conv2d(down1_maxpool2d_0_output, down1_maxpool_conv_1_double_conv_0_weight,
           NULL, down1_conv_0_output, 1, 1, 1, false);
    BatchNorm2d(down1_conv_0_output, down1_maxpool_conv_1_double_conv_1_weight,
                down1_maxpool_conv_1_double_conv_1_bias,
                down1_batchnorm_0_running_mean, down1_batchnorm_0_running_var,
                down1_batchnorm_0_output, 1e-5, 0.1);
    ReLU(down1_batchnorm_0_output);
    Conv2d(down1_batchnorm_0_output, down1_maxpool_conv_1_double_conv_3_weight,
           NULL, down1_conv_1_output, 1, 1, 1, false);
    BatchNorm2d(down1_conv_1_output, down1_maxpool_conv_1_double_conv_4_weight,
                down1_maxpool_conv_1_double_conv_4_bias,
                down1_batchnorm_1_running_mean, down1_batchnorm_1_running_var,
                down1_batchnorm_1_output, 1e-5, 0.1);
    ReLU(down1_batchnorm_1_output);

    // down2(128, 256)
    MaxPool2d(down1_batchnorm_1_output, down2_maxpool2d_0_output);
    Conv2d(down2_maxpool2d_0_output, down2_maxpool_conv_1_double_conv_0_weight,
           NULL, down2_conv_0_output, 1, 1, 1, false);
    BatchNorm2d(down2_conv_0_output, down2_maxpool_conv_1_double_conv_1_weight,
                down2_maxpool_conv_1_double_conv_1_bias,
                down2_batchnorm_0_running_mean, down2_batchnorm_0_running_var,
                down2_batchnorm_0_output, 1e-5, 0.1);
    ReLU(down2_batchnorm_0_output);
    Conv2d(down2_batchnorm_0_output, down2_maxpool_conv_1_double_conv_3_weight,
           NULL, down2_conv_1_output, 1, 1, 1, false);
    BatchNorm2d(down2_conv_1_output, down2_maxpool_conv_1_double_conv_4_weight,
                down2_maxpool_conv_1_double_conv_4_bias,
                down2_batchnorm_1_running_mean, down2_batchnorm_1_running_var,
                down2_batchnorm_1_output, 1e-5, 0.1);
    ReLU(down2_batchnorm_1_output);

    // up1(256, 128), (up2_concat_0_output, down1_batchnorm_1_output)
    ConvTranspose2d(down2_batchnorm_1_output, up1_up_weight, up1_up_bias,
                    up1_convt_0_output, 2, 0);
    Concat(up1_convt_0_output, down1_batchnorm_1_output, up1_concat_0_output);
    Conv2d(up1_concat_0_output, up1_conv_double_conv_0_weight, NULL,
           up1_conv_0_output, 1, 1, 1, false);
    BatchNorm2d(up1_conv_0_output, up1_conv_double_conv_1_weight,
                up1_conv_double_conv_1_bias, up1_batchnorm_0_running_mean,
                up1_batchnorm_0_running_var, up1_batchnorm_0_output, 1e-5, 0.1);
    ReLU(up1_batchnorm_0_output);
    Conv2d(up1_batchnorm_0_output, up1_conv_double_conv_3_weight, NULL,
           up1_conv_1_output, 1, 1, 1, false);
    BatchNorm2d(up1_conv_1_output, up1_conv_double_conv_4_weight,
                up1_conv_double_conv_4_bias, up1_batchnorm_1_running_mean,
                up1_batchnorm_1_running_var, up1_batchnorm_1_output, 1e-5, 0.1);
    ReLU(up1_batchnorm_1_output);

    // up2(128, 64), (up1_concat_0_output, inc_batchnorm_1_output)
    ConvTranspose2d(up1_batchnorm_1_output, up2_up_weight, up2_up_bias,
                    up2_convt_0_output, 2, 0);
    Concat(up2_convt_0_output, inc_batchnorm_1_output, up2_concat_0_output);
    Conv2d(up2_concat_0_output, up2_conv_double_conv_0_weight, NULL,
           up2_conv_0_output, 1, 1, 1, false);
    BatchNorm2d(up2_conv_0_output, up2_conv_double_conv_1_weight,
                up2_conv_double_conv_1_bias, up2_batchnorm_0_running_mean,
                up2_batchnorm_0_running_var, up2_batchnorm_0_output, 1e-5, 0.1);
    ReLU(up2_batchnorm_0_output);
    Conv2d(up2_batchnorm_0_output, up2_conv_double_conv_3_weight, NULL,
           up2_conv_1_output, 1, 1, 1, false);
    BatchNorm2d(up2_conv_1_output, up2_conv_double_conv_4_weight,
                up2_conv_double_conv_4_bias, up2_batchnorm_1_running_mean,
                up2_batchnorm_1_running_var, up2_batchnorm_1_output, 1e-5, 0.1);
    ReLU(up2_batchnorm_1_output);

    // outc(64, 2)
    Conv2d(up2_batchnorm_1_output, outc_conv_weight, outc_conv_bias, output, 1,
           0, 1, true);

    memcpy(outputN->buf + (idx * 1 * 2 * 128 * 191), output->buf,
           sizeof(float) * (1 * 2 * 128 * 191));
  }
}

/* Operations */

/*
 * Convolution
 * input shape = (N, C, H, W)
 * weight shape = (K, C, R, S)
 * bias shape = (K)
 * output shape = (N, K, OH, OW)
 *   where OH = (H + 2 * pad - dilation * (R - 1) - 1) / stride + 1,
 *         OW = (W + 2 * pad - dilation * (S - 1) - 1) / stride + 1
 */
void Conv2d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride, int pad, int dilation, bool has_bias) {
  int C = input->shape[1], H = input->shape[2], W = input->shape[3];
  int K = weight->shape[0], R = weight->shape[2], S = weight->shape[3];
  int OH = output->shape[2], OW = output->shape[3];
  printf("Kernel Size : %d  filter Width : %d   filter Height: %d   \n",K, R, S);
  CHECK_ERROR(OH == (H + 2 * pad - dilation * (R - 1) - 1) / stride + 1,
              "[Conv2d] Output height mismatch");
  CHECK_ERROR(OW == (W + 2 * pad - dilation * (S - 1) - 1) / stride + 1,
              "[Conv2d] Output width mismatch");
  CHECK_ERROR(weight->shape[1] == C && (!has_bias || bias->shape[0] == K) &&
                  output->shape[1] == K,
              "[Conv2d] Channel size mismatch");

#ifdef TEST
#pragma omp parallel for
#endif
  for (int k = 0; k < K; ++k) {              // filter's channel
    for (int oh = 0; oh < OH; ++oh) {        // output hieght
      for (int ow = 0; ow < OW; ++ow) {      // output width
        float o = has_bias ? bias->buf[k] : 0;  
        for (int c = 0; c < C; ++c) {        // input's channel
          for (int r = 0; r < R; ++r) {      // filter's hieght
            for (int s = 0; s < S; ++s) {    // filter's width
              int h = oh * stride - pad + r * dilation;
              int w = ow * stride - pad + s * dilation;
              if (h < 0 || h >= H || w < 0 || w >= W) continue;
              float i = input->buf[c * H * W + h * W + w];
              float f = weight->buf[k * C * R * S + c * R * S + r * S + s];
              o += i * f;
            }
          }
        }
        output->buf[k * OH * OW + oh * OW + ow] = o;
      }
    }
  }
}

/*
 * ReLU
 * input shape = (N, C, H, W)
 * output shape = (N, C, H, W)
 * Formula: y = max(x, 0)
 */
void ReLU(Tensor *inout) {
  int C = inout->shape[1], H = inout->shape[2], W = inout->shape[3];

  for (int c = 0; c < C; ++c) {
    for (int h = 0; h < H; ++h) {
      for (int w = 0; w < W; ++w) {
        int idx = c * H * W + h * W + w;
        inout->buf[idx] = inout->buf[idx] > 0 ? inout->buf[idx] : 0;
      }
    }
  }
}

/*
 * Batch Normaliztion
 * input shape = (N, C, H, W)
 * gamma shape = (C)
 * beta shape = (C)
 * output shape = (N, C, H, W)
 */
void BatchNorm2d(Tensor *input, Tensor *gamma, Tensor *beta,
                 Tensor *running_mean, Tensor *running_var, Tensor *output,
                 const float eps, const float momentum) {
  int N = input->shape[0], C = input->shape[1], H = input->shape[2],
      W = input->shape[3];

  CHECK_ERROR(gamma->shape[0] == C && beta->shape[0] == C,
              "[BatchNorm2d] gamma, beta shape mismatch");
  CHECK_ERROR(
      output->shape[1] == C && output->shape[2] == H && output->shape[3] == W,
      "[BatchNorm2d] Output shape mismatch");

  for (int c = 0; c < C; ++c) {
    for (int n = 0; n < N; ++n) {
      for (int h = 0; h < H; ++h) {
        for (int w = 0; w < W; ++w) {
          float mean = running_mean->buf[c];
          float variance = running_var->buf[c];
          float x = input->buf[n * C * H * W + c * H * W + h * W + w];
          float x_hat = (x - mean) / sqrt(variance + eps);
          output->buf[n * C * H * W + c * H * W + h * W + w] =
              gamma->buf[c] * x_hat + beta->buf[c];
        }
      }
    }
  }
}

/*
 * Transposed convolution
 * input shape = (N, C, H, W)
 * weight shape = (C, K, R, S)
 * bias shape = (K)
 * output shape = (N, K, OH, OW)
 *   where OH = (H - 1) * stride - 2 * pad + R
 *         OW = (W - 1) * stride - 2 * pad + S
 */
void ConvTranspose2d(Tensor *input, Tensor *weight, Tensor *bias,
                     Tensor *output, int stride, int pad) {
  int C = input->shape[1], H = input->shape[2], W = input->shape[3];
  int K = weight->shape[1], R = weight->shape[2], S = weight->shape[3];
  int OH = output->shape[2], OW = output->shape[3];

  CHECK_ERROR(OH == (H - 1) * stride - 2 * pad + R,
              "[ConvT2d] Output height mismatch");
  CHECK_ERROR(OW == (W - 1) * stride - 2 * pad + S,
              "[ConvT2d] Output width mismatch");
  CHECK_ERROR(
      weight->shape[0] == C && bias->shape[0] == K && output->shape[1] == K,
      "[ConvT2d] Channel size mismatch");

  for (int k = 0; k < K; ++k) {
    for (int oh = 0; oh < OH; ++oh) {
      for (int ow = 0; ow < OW; ++ow) {
        float o = bias->buf[k];
        for (int c = 0; c < C; ++c) {
          for (int r = 0; r < R; ++r) {
            for (int s = 0; s < S; ++s) {
              if ((oh + pad - r) % stride != 0) continue;
              if ((ow + pad - s) % stride != 0) continue;
              int h = (oh + pad - r) / stride;
              int w = (ow + pad - s) / stride;
              if (h < 0 || h >= H || w < 0 || w >= W) continue;
              float i = input->buf[c * H * W + h * W + w];
              float f = weight->buf[c * K * R * S + k * R * S + r * S + s];
              o += i * f;
            }
          }
        }
        output->buf[k * OH * OW + oh * OW + ow] = o;
      }
    }
  }
}

float max4(float in0, float in1, float in2, float in3) {
  float max = in0;

  if (in1 > max) max = in1;
  if (in2 > max) max = in2;
  if (in3 > max) max = in3;
  return max;
}

/*
 * MaxPool2d
 * input shape = (N, C, H, W)
 * output shape = (N, OC, OH, OW)
 *   where OH = H / 2
 *         OW = W / 2
 */
void MaxPool2d(Tensor *input, Tensor *output) {
  int C = input->shape[1], H = input->shape[2], W = input->shape[3];
  int OC = output->shape[1], OH = output->shape[2], OW = output->shape[3];

  CHECK_ERROR(OW == W / 2, "[MaxPool2d] Output width mismatch");
  CHECK_ERROR(OH == H / 2, "[MaxPool2d] Output height mismatch");
  CHECK_ERROR(OC == C, "[MaxPool2d] Output channel mismatch");

  for (int oc = 0; oc < OC; ++oc) {
    for (int oh = 0; oh < OH; ++oh) {
      for (int ow = 0; ow < OW; ++ow) {
        float in0 = input->buf[oc * H * W + 2 * oh * W + 2 * ow];
        float in1 = input->buf[oc * H * W + 2 * oh * W + 2 * ow + 1];
        float in2 = input->buf[oc * H * W + (2 * oh + 1) * W + 2 * ow];
        float in3 = input->buf[oc * H * W + (2 * oh + 1) * W + 2 * ow + 1];
        output->buf[oc * OH * OW + oh * OW + ow] = max4(in0, in1, in2, in3);
      }
    }
  }
}

/*
 * Concat
 * input1 shape = (N, C1, H1, W1)
 * input2 shape = (N, C2, H2, W2)
 * output shape = (N, OC, OH, OW)
 *   where OH = H2, H1
 *         OW = W2 = W1 + 1
 */
void Concat(Tensor *input1, Tensor *input2, Tensor *output) {
  int C1 = input1->shape[1], H1 = input1->shape[2], W1 = input1->shape[3];
  int C2 = input2->shape[1], H2 = input2->shape[2], W2 = input2->shape[3];
  int OC = output->shape[1], OH = output->shape[2], OW = output->shape[3];

  CHECK_ERROR(OC == C1 * 2 && OC == C2 * 2, "[Concat] Output channel mismatch");
  CHECK_ERROR(OW == W1 + 1 && OW == W2, "[Concat] Output width mismatch");
  CHECK_ERROR(OH == H1 && OH == H2, "[Concat] Output height mismatch");

  for (int oc = 0; oc < OC / 2; ++oc) {
    for (int oh = 0; oh < OH; ++oh) {
      for (int ow = 0; ow < OW; ++ow) {
        output->buf[oc * OH * OW + oh * OW + ow] =
            input2->buf[oc * OH * OW + oh * OW + ow];
      }
    }
  }

  for (int oc = OC / 2; oc < OC; ++oc) {
    for (int oh = 0; oh < OH; ++oh) {
      for (int ow = 0; ow < OW; ++ow) {
        if (ow == OW - 1)
          output->buf[oc * OH * OW + oh * OW + ow] = 0.0;  // zero padding
        else
          output->buf[oc * OH * OW + oh * OW + ow] =
              input1->buf[(oc - OC / 2) * H1 * W1 + oh * W1 + ow];
      }
    }
  }
}

/*
 * uNet_initialize
 * Initialize the model. Do input-independent job here.
 */
void uNet_initialize(int N, char *parameter_fname) {
  size_t parameter_binary_size = 0;
  float *parameter =
      (float *) read_binary(parameter_fname, &parameter_binary_size);

  // Parameters
  inc_double_conv_0_weight = new Tensor({64, 3, 3, 3}, parameter + OFFSET0);
  inc_double_conv_1_weight = new Tensor({64}, parameter + OFFSET1);
  inc_double_conv_1_bias = new Tensor({64}, parameter + OFFSET2);
  inc_double_conv_3_weight = new Tensor({64, 64, 3, 3}, parameter + OFFSET3);
  inc_double_conv_4_weight = new Tensor({64}, parameter + OFFSET4);
  inc_double_conv_4_bias = new Tensor({64}, parameter + OFFSET5);
  down1_maxpool_conv_1_double_conv_0_weight =
      new Tensor({128, 64, 3, 3}, parameter + OFFSET6);
  down1_maxpool_conv_1_double_conv_1_weight =
      new Tensor({128}, parameter + OFFSET7);
  down1_maxpool_conv_1_double_conv_1_bias =
      new Tensor({128}, parameter + OFFSET8);
  down1_maxpool_conv_1_double_conv_3_weight =
      new Tensor({128, 128, 3, 3}, parameter + OFFSET9);
  down1_maxpool_conv_1_double_conv_4_weight =
      new Tensor({128}, parameter + OFFSET10);
  down1_maxpool_conv_1_double_conv_4_bias =
      new Tensor({128}, parameter + OFFSET11);
  down2_maxpool_conv_1_double_conv_0_weight =
      new Tensor({256, 128, 3, 3}, parameter + OFFSET12);
  down2_maxpool_conv_1_double_conv_1_weight =
      new Tensor({256}, parameter + OFFSET13);
  down2_maxpool_conv_1_double_conv_1_bias =
      new Tensor({256}, parameter + OFFSET14);
  down2_maxpool_conv_1_double_conv_3_weight =
      new Tensor({256, 256, 3, 3}, parameter + OFFSET15);
  down2_maxpool_conv_1_double_conv_4_weight =
      new Tensor({256}, parameter + OFFSET16);
  down2_maxpool_conv_1_double_conv_4_bias =
      new Tensor({256}, parameter + OFFSET17);
  up1_up_weight = new Tensor({256, 128, 2, 2}, parameter + OFFSET18);
  up1_up_bias = new Tensor({128}, parameter + OFFSET19);
  up1_conv_double_conv_0_weight =
      new Tensor({128, 256, 3, 3}, parameter + OFFSET20);
  up1_conv_double_conv_1_weight = new Tensor({128}, parameter + OFFSET21);
  up1_conv_double_conv_1_bias = new Tensor({128}, parameter + OFFSET22);
  up1_conv_double_conv_3_weight =
      new Tensor({128, 128, 3, 3}, parameter + OFFSET23);
  up1_conv_double_conv_4_weight = new Tensor({128}, parameter + OFFSET24);
  up1_conv_double_conv_4_bias = new Tensor({128}, parameter + OFFSET25);
  up2_up_weight = new Tensor({128, 64, 2, 2}, parameter + OFFSET26);
  up2_up_bias = new Tensor({64}, parameter + OFFSET27);
  up2_conv_double_conv_0_weight =
      new Tensor({64, 128, 3, 3}, parameter + OFFSET28);
  up2_conv_double_conv_1_weight = new Tensor({64}, parameter + OFFSET29);
  up2_conv_double_conv_1_bias = new Tensor({64}, parameter + OFFSET30);
  up2_conv_double_conv_3_weight =
      new Tensor({64, 64, 3, 3}, parameter + OFFSET31);
  up2_conv_double_conv_4_weight = new Tensor({64}, parameter + OFFSET32);
  up2_conv_double_conv_4_bias = new Tensor({64}, parameter + OFFSET33);
  outc_conv_weight = new Tensor({2, 64, 1, 1}, parameter + OFFSET34);
  outc_conv_bias = new Tensor({2}, parameter + OFFSET35);
  inc_batchnorm_0_running_mean = new Tensor({64}, parameter + OFFSET36);
  inc_batchnorm_0_running_var = new Tensor({64}, parameter + OFFSET37);
  inc_batchnorm_1_running_mean = new Tensor({64}, parameter + OFFSET38);
  inc_batchnorm_1_running_var = new Tensor({64}, parameter + OFFSET39);
  down1_batchnorm_0_running_mean = new Tensor({128}, parameter + OFFSET40);
  down1_batchnorm_0_running_var = new Tensor({128}, parameter + OFFSET41);
  down1_batchnorm_1_running_mean = new Tensor({128}, parameter + OFFSET42);
  down1_batchnorm_1_running_var = new Tensor({128}, parameter + OFFSET43);
  down2_batchnorm_0_running_mean = new Tensor({256}, parameter + OFFSET44);
  down2_batchnorm_0_running_var = new Tensor({256}, parameter + OFFSET45);
  down2_batchnorm_1_running_mean = new Tensor({256}, parameter + OFFSET46);
  down2_batchnorm_1_running_var = new Tensor({256}, parameter + OFFSET47);
  up1_batchnorm_0_running_mean = new Tensor({128}, parameter + OFFSET48);
  up1_batchnorm_0_running_var = new Tensor({128}, parameter + OFFSET49);
  up1_batchnorm_1_running_mean = new Tensor({128}, parameter + OFFSET50);
  up1_batchnorm_1_running_var = new Tensor({128}, parameter + OFFSET51);
  up2_batchnorm_0_running_mean = new Tensor({64}, parameter + OFFSET52);
  up2_batchnorm_0_running_var = new Tensor({64}, parameter + OFFSET53);
  up2_batchnorm_1_running_mean = new Tensor({64}, parameter + OFFSET54);
  up2_batchnorm_1_running_var = new Tensor({64}, parameter + OFFSET55);

  // Activations
  inc_conv_0_output = new Tensor({1, 64, 128, 191});
  inc_batchnorm_0_output = new Tensor({1, 64, 128, 191});
  inc_conv_1_output = new Tensor({1, 64, 128, 191});
  inc_batchnorm_1_output = new Tensor({1, 64, 128, 191});

  down1_maxpool2d_0_output = new Tensor({1, 64, 64, 95});
  down1_conv_0_output = new Tensor({1, 128, 64, 95});
  down1_batchnorm_0_output = new Tensor({1, 128, 64, 95});
  down1_conv_1_output = new Tensor({1, 128, 64, 95});
  down1_batchnorm_1_output = new Tensor({1, 128, 64, 95});

  down2_maxpool2d_0_output = new Tensor({1, 128, 32, 47});
  down2_conv_0_output = new Tensor({1, 256, 32, 47});
  down2_batchnorm_0_output = new Tensor({1, 256, 32, 47});
  down2_conv_1_output = new Tensor({1, 256, 32, 47});
  down2_batchnorm_1_output = new Tensor({1, 256, 32, 47});

  up1_convt_0_output = new Tensor({1, 128, 64, 94});
  up1_concat_0_output = new Tensor({1, 256, 64, 95});
  up1_conv_0_output = new Tensor({1, 128, 64, 95});
  up1_batchnorm_0_output = new Tensor({1, 128, 64, 95});
  up1_conv_1_output = new Tensor({1, 128, 64, 95});
  up1_batchnorm_1_output = new Tensor({1, 128, 64, 95});

  up2_convt_0_output = new Tensor({1, 64, 128, 190});
  up2_concat_0_output = new Tensor({1, 128, 128, 191});
  up2_conv_0_output = new Tensor({1, 64, 128, 191});
  up2_batchnorm_0_output = new Tensor({1, 64, 128, 191});
  up2_conv_1_output = new Tensor({1, 64, 128, 191});
  up2_batchnorm_1_output = new Tensor({1, 64, 128, 191});
  outc_conv_0_output = new Tensor({1, 2, 128, 191});
}

/*
 * uNet_finalize
 * Finalize the model.
 */
void uNet_finalize() {
  // delete parameters
  delete inc_double_conv_0_weight;
  delete inc_double_conv_1_weight;
  delete inc_double_conv_1_bias;
  delete inc_double_conv_3_weight;
  delete inc_double_conv_4_weight;
  delete inc_double_conv_4_bias;
  delete down1_maxpool_conv_1_double_conv_0_weight;
  delete down1_maxpool_conv_1_double_conv_1_weight;
  delete down1_maxpool_conv_1_double_conv_1_bias;
  delete down1_maxpool_conv_1_double_conv_3_weight;
  delete down1_maxpool_conv_1_double_conv_4_weight;
  delete down1_maxpool_conv_1_double_conv_4_bias;
  delete down2_maxpool_conv_1_double_conv_0_weight;
  delete down2_maxpool_conv_1_double_conv_1_weight;
  delete down2_maxpool_conv_1_double_conv_1_bias;
  delete down2_maxpool_conv_1_double_conv_3_weight;
  delete down2_maxpool_conv_1_double_conv_4_weight;
  delete down2_maxpool_conv_1_double_conv_4_bias;
  delete up1_up_weight;
  delete up1_up_bias;
  delete up1_conv_double_conv_0_weight;
  delete up1_conv_double_conv_1_weight;
  delete up1_conv_double_conv_1_bias;
  delete up1_conv_double_conv_3_weight;
  delete up1_conv_double_conv_4_weight;
  delete up1_conv_double_conv_4_bias;
  delete up2_up_weight;
  delete up2_up_bias;
  delete up2_conv_double_conv_0_weight;
  delete up2_conv_double_conv_1_weight;
  delete up2_conv_double_conv_1_bias;
  delete up2_conv_double_conv_3_weight;
  delete up2_conv_double_conv_4_weight;
  delete up2_conv_double_conv_4_bias;
  delete outc_conv_weight;
  delete outc_conv_bias;
  delete inc_batchnorm_0_running_mean;
  delete inc_batchnorm_0_running_var;
  delete down1_batchnorm_0_running_mean;
  delete down1_batchnorm_0_running_var;
  delete down2_batchnorm_0_running_mean;
  delete down2_batchnorm_0_running_var;
  delete up1_batchnorm_0_running_mean;
  delete up1_batchnorm_0_running_var;
  delete up2_batchnorm_0_running_mean;
  delete up2_batchnorm_0_running_var;
  delete inc_batchnorm_1_running_mean;
  delete inc_batchnorm_1_running_var;
  delete down1_batchnorm_1_running_mean;
  delete down1_batchnorm_1_running_var;
  delete down2_batchnorm_1_running_mean;
  delete down2_batchnorm_1_running_var;
  delete up1_batchnorm_1_running_mean;
  delete up1_batchnorm_1_running_var;
  delete up2_batchnorm_1_running_mean;
  delete up2_batchnorm_1_running_var;

  // delete activations
  delete inc_conv_0_output;
  delete inc_batchnorm_0_output;
  delete inc_conv_1_output;
  delete inc_batchnorm_1_output;
  delete down1_maxpool2d_0_output;
  delete down1_conv_0_output;
  delete down1_batchnorm_0_output;
  delete down1_conv_1_output;
  delete down1_batchnorm_1_output;
  delete down2_maxpool2d_0_output;
  delete down2_conv_0_output;
  delete down2_batchnorm_0_output;
  delete down2_conv_1_output;
  delete down2_batchnorm_1_output;
  delete up1_convt_0_output;
  delete up1_concat_0_output;
  delete up1_conv_0_output;
  delete up1_batchnorm_0_output;
  delete up1_conv_1_output;
  delete up1_batchnorm_1_output;
  delete up2_convt_0_output;
  delete up2_concat_0_output;
  delete up2_conv_0_output;
  delete up2_batchnorm_0_output;
  delete up2_conv_1_output;
  delete up2_batchnorm_1_output;
  delete outc_conv_0_output;
}
